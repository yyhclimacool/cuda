#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void vec_add(int n, float *da, float *db, float *dc) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    dc[idx] = da[idx] + db[idx] + 1.0;
  }
  if (idx / 1024 == 0) {
    printf("idx[%d]", idx);
  }
}

int main() {
  int n = 1024 * 1024;
  int nbytes = n * sizeof(float);
  float *a = 0, *b = 0, *c = 0;
  float *da = 0, *db = 0, *dc = 0;

  a = (float *)malloc(nbytes);
  b = (float *)malloc(nbytes);
  c = (float *)malloc(nbytes);

  srand(time(0));

  for (int i = 0; i < n; ++i) {
    a[i] = (float)(rand()) / (float)(rand());
    b[i] = (float)(rand()) / (float)(rand());
    c[i] = 0;
  }

  hipMalloc((void **)&da, nbytes);
  hipMalloc((void **)&db, nbytes);

  hipMemcpy(da, a, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(db, b, nbytes, hipMemcpyHostToDevice);

  vec_add<<<(n + 255) / 256, 256>>>(n, da, db, dc);

  hipMemcpy(c, dc, nbytes, hipMemcpyDeviceToHost);

  double result = 0;
  for (int i = 0; i < n; ++i) {
    result += c[i];
  }
  std::cout << "result[" << result << "]" << std::endl;

  free(a);
  free(b);
  free(c);

  hipFree(da);
  hipFree(db);
  hipFree(dc);
}
